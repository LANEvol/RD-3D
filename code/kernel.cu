#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <set>
#include <vector>
#include <algorithm>
#include <string>
#include <cstdlib>
#include <ctime>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <hip/hip_runtime.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>
//#include <timer.h>               // timing functions

// CUDA helper functions
//#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check 
#include <hiprand.h>
#include <map>
#include <utility>



using namespace std;

GLuint gl_u, gl_v, gl_points;

double *d_u_old, *d_u_new, *d_v_old, *d_v_new, *d_w_old, *d_w_new;
float  *border, *d_border, *d_z, *z, *Deff, *d_Deff;

double *u_new, *v_new, *w_new, *u_old, *v_old, *w_old, *centers, *pars;

int  *closest_el, *neighbours;
bool *initialColors;
bool periodic, constantDiffusion, correction, growth, incrementE;

map<int, vector<int> >  contributions;
map<int, vector<int> >  nodesPerHexa;

int nodesSize, a, b, c, centersSize, nodesToSave, networkType;
int  saveFrequency, iteration, saveIteration, quasiStaticStep;
int stop, saving;
double h, dt, Du, Dv, Dw, U, V, W, cu, cV, cw, c1, c2, c3, c4, c5, c6, c7, c8, c9, error, hz,P, bt, border_thickness_in_elements,sigma, minDiffusionFactor;
double initialGreen[3];
double initialBlack[3];
double initialUniform[3];
double sx, sy;
char saveFolder[200];


dim3 DimBlock;
dim3 DimGrid;
dim3 DimBlockSim;
dim3 DimGridSim;



__constant__ int dimensions[3];
__constant__ double parameters3var[20];
texture <float, 1, hipReadModeElementType> borderTexture;
texture <float, 1, hipReadModeElementType> zTexture;
texture <float, 1, hipReadModeElementType> DeffTexture;



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__device__ double reaction_u(double u1, double v1, double w1)
{
	double U = parameters3var[2] * v1 + parameters3var[3] * w1 + parameters3var[4];
	if(U > parameters3var[17])
	{
		return parameters3var[17];
	}
	else if(U < 0)
	{
		return 0;
	}
	return U;
}

__device__ double reaction_v(double u1, double v1, double w1)
{
	double U = parameters3var[5] * u1 + parameters3var[6] * w1 + parameters3var[7];
	if(U > parameters3var[18])
	{
		return parameters3var[18];
	}
	else if(U < 0)
	{
		return 0;
	}
	return U;
}

__device__ double reaction_w(double u1, double v1, double w1)
{
	double U = parameters3var[8] * u1 + parameters3var[9] * v1 + parameters3var[10];
	if(U > parameters3var[19])
	{
		return parameters3var[19];
	}
	else if(U < 0)
	{
		return 0;
	}
	return U;
}



__global__ void mainFunction3D3varBorder_P(double *u_old, double *u_new, double *v_old, double *v_new, double *w_old, double *w_new)
{
	__shared__ double u[16][16][4];
	__shared__ double v[16][16][4];
	__shared__ double w[16][16][4];
	int myIndX = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
	int myIndY = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
	int myIndZ = blockIdx.z * (blockDim.z - 2) + threadIdx.z - 1;
	int myIndXLocal = threadIdx.x;
	int myIndYLocal = threadIdx.y;
	int myIndZLocal = threadIdx.z;
	int index = 0;
	bool go = false;


	if(myIndX < (dimensions[0] + 1) && myIndY < (dimensions[1] + 1) && myIndZ < (dimensions[2] + 1))
	{
		if(myIndX >=0 && myIndX <= dimensions[0] - 1  && myIndY >= 0 && myIndY <= dimensions[1] - 1 && myIndZ >= 0 && myIndZ <= dimensions[2] - 1)
		{
			go = true;
		}
		if(myIndX < 0) myIndX = dimensions[0] - 1;
		if(myIndY < 0) myIndY = dimensions[1] - 1;
		if(myIndZ < 0) myIndZ = 0; 
		if(myIndX == dimensions[0]) myIndX = 0;
		if(myIndY == dimensions[1]) myIndY  = 0;
		if(myIndZ == dimensions[2]) myIndZ  = dimensions[2] - 1;
		//global index in 1D array
		index = myIndZ * (dimensions[0] * dimensions[1]) + myIndX * dimensions[1] + myIndY;

		u[myIndXLocal][myIndYLocal][myIndZLocal] = u_old[index];
		v[myIndXLocal][myIndYLocal][myIndZLocal] = v_old[index];
		w[myIndXLocal][myIndYLocal][myIndZLocal] = w_old[index];
	}
	//solve for the indices out of domain
	__syncthreads();
	if(go && myIndXLocal > 0 && myIndXLocal < 15 && myIndYLocal > 0 && myIndYLocal < 15 && myIndZLocal > 0 && myIndZLocal < 3)
	{

		float xl = tex1Dfetch(borderTexture, 6 * index);
		float xr = tex1Dfetch(borderTexture, 6 * index + 1);
		float yl = tex1Dfetch(borderTexture, 6 * index + 2);
		float yr = tex1Dfetch(borderTexture, 6 * index + 3);
		float zl = tex1Dfetch(borderTexture, 6 * index + 4);
		float zr = tex1Dfetch(borderTexture, 6 * index + 5);
		
		//check if the node has to be simulated
		if(xl < -1) return;

		double Laplacian_u = 0;
		double Laplacian_v = 0;
		double Laplacian_w = 0;
		
		Laplacian_u += xl * (u[myIndXLocal - 1][myIndYLocal][myIndZLocal] - u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v += xl * (v[myIndXLocal - 1][myIndYLocal][myIndZLocal] - v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w += xl * (w[myIndXLocal - 1][myIndYLocal][myIndZLocal] - w[myIndXLocal][myIndYLocal][myIndZLocal]);	
		
		Laplacian_u += xr * (u[myIndXLocal + 1][myIndYLocal][myIndZLocal] - u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v += xr * (v[myIndXLocal + 1][myIndYLocal][myIndZLocal] - v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w += xr * (w[myIndXLocal + 1][myIndYLocal][myIndZLocal] - w[myIndXLocal][myIndYLocal][myIndZLocal]);		
		
		Laplacian_u += yl * (u[myIndXLocal][myIndYLocal - 1][myIndZLocal] - u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v += yl * (v[myIndXLocal][myIndYLocal - 1][myIndZLocal] - v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w += yl * (w[myIndXLocal][myIndYLocal - 1][myIndZLocal] - w[myIndXLocal][myIndYLocal][myIndZLocal]);		
		
		Laplacian_u += yr * (u[myIndXLocal][myIndYLocal + 1][myIndZLocal] - u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v += yr * (v[myIndXLocal][myIndYLocal + 1][myIndZLocal] - v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w += yr * (w[myIndXLocal][myIndYLocal + 1][myIndZLocal] - w[myIndXLocal][myIndYLocal][myIndZLocal]);			
	
		Laplacian_u += zl * (u[myIndXLocal][myIndYLocal][myIndZLocal - 1] - u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v += zl * (v[myIndXLocal][myIndYLocal][myIndZLocal - 1] - v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w += zl * (w[myIndXLocal][myIndYLocal][myIndZLocal - 1] - w[myIndXLocal][myIndYLocal][myIndZLocal]);

		Laplacian_u += zr * (u[myIndXLocal][myIndYLocal][myIndZLocal + 1] - u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v += zr * (v[myIndXLocal][myIndYLocal][myIndZLocal + 1] - v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w += zr * (w[myIndXLocal][myIndYLocal][myIndZLocal + 1] - w[myIndXLocal][myIndYLocal][myIndZLocal]);
		
		Laplacian_u = u[myIndXLocal][myIndYLocal][myIndZLocal] + parameters3var[1] * (Laplacian_u * parameters3var[14]/parameters3var[0] + reaction_u(u[myIndXLocal][myIndYLocal][myIndZLocal], v[myIndXLocal][myIndYLocal][myIndZLocal], w[myIndXLocal][myIndYLocal][myIndZLocal]) - parameters3var[11] * u[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_v = v[myIndXLocal][myIndYLocal][myIndZLocal] + parameters3var[1] * (Laplacian_v * parameters3var[15]/parameters3var[0] + reaction_v(u[myIndXLocal][myIndYLocal][myIndZLocal], v[myIndXLocal][myIndYLocal][myIndZLocal], w[myIndXLocal][myIndYLocal][myIndZLocal]) - parameters3var[12] * v[myIndXLocal][myIndYLocal][myIndZLocal]);
		Laplacian_w = w[myIndXLocal][myIndYLocal][myIndZLocal] + parameters3var[1] * (Laplacian_w * parameters3var[16]/parameters3var[0] + reaction_w(u[myIndXLocal][myIndYLocal][myIndZLocal], v[myIndXLocal][myIndYLocal][myIndZLocal], w[myIndXLocal][myIndYLocal][myIndZLocal]) - parameters3var[13] * w[myIndXLocal][myIndYLocal][myIndZLocal]);

		u_new[index] = Laplacian_u > 0 ? Laplacian_u : 0;
		v_new[index] = Laplacian_v > 0 ? Laplacian_v : 0;
		w_new[index] = Laplacian_w > 0 ? Laplacian_w : 0;
	}
}

//it works in 2D only for a squared lattice, otherwise call 3D method
__global__ void mainFunction2D3varBorder_P(double *u_old, double *u_new, double *v_old, double *v_new, double *w_old, double *w_new)
{
	__shared__ double u[32][32];
	__shared__ double v[32][32];
	__shared__ double w[32][32];
	int myIndX = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
	int myIndY = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
	int myIndXLocal = threadIdx.x;
	int myIndYLocal = threadIdx.y;
	int index = 0;
	bool go = false;


	if(myIndX < (dimensions[0] + 1) && myIndY < (dimensions[1] + 1))
	{
		if(myIndX >=0 && myIndX <= dimensions[0] - 1  && myIndY >= 0 && myIndY <= dimensions[1] - 1)
		{
			go = true;
		}
		if(myIndX < 0) myIndX = 0;
		if(myIndY < 0) myIndY = 0;
		if(myIndX == dimensions[0]) myIndX = dimensions[0] - 1;
		if(myIndY == dimensions[1]) myIndY  = dimensions[1] - 1;
		//global index in 1D array
		index = myIndX * dimensions[1] + myIndY;

		u[myIndXLocal][myIndYLocal] = u_old[index];
		v[myIndXLocal][myIndYLocal] = v_old[index];
		w[myIndXLocal][myIndYLocal] = w_old[index];
	}
	//solve for the indices out of domain
	__syncthreads();
	if(go && myIndXLocal > 0 && myIndXLocal < 31 && myIndYLocal > 0 && myIndYLocal < 31)
	{


		float xl = tex1Dfetch(borderTexture, 6 * index);
		float xr = tex1Dfetch(borderTexture, 6 * index + 1);
		float yl = tex1Dfetch(borderTexture, 6 * index + 2);
		float yr = tex1Dfetch(borderTexture, 6 * index + 3);
		double Laplacian_u = 0;
		double Laplacian_v = 0;
		double Laplacian_w = 0;
		
		if(xl < -1) return;
		
		Laplacian_u += xl * (u[myIndXLocal - 1][myIndYLocal] - u[myIndXLocal][myIndYLocal]);
		Laplacian_v += xl * (v[myIndXLocal - 1][myIndYLocal] - v[myIndXLocal][myIndYLocal]);
		Laplacian_w += xl * (w[myIndXLocal - 1][myIndYLocal] - w[myIndXLocal][myIndYLocal]);	
		
		Laplacian_u += xr * (u[myIndXLocal + 1][myIndYLocal] - u[myIndXLocal][myIndYLocal]);
		Laplacian_v += xr * (v[myIndXLocal + 1][myIndYLocal] - v[myIndXLocal][myIndYLocal]);
		Laplacian_w += xr * (w[myIndXLocal + 1][myIndYLocal] - w[myIndXLocal][myIndYLocal]);		
		
		Laplacian_u += yl * (u[myIndXLocal][myIndYLocal - 1] - u[myIndXLocal][myIndYLocal]);
		Laplacian_v += yl * (v[myIndXLocal][myIndYLocal - 1] - v[myIndXLocal][myIndYLocal]);
		Laplacian_w += yl * (w[myIndXLocal][myIndYLocal - 1] - w[myIndXLocal][myIndYLocal]);		
		
		Laplacian_u += yr * (u[myIndXLocal][myIndYLocal + 1] - u[myIndXLocal][myIndYLocal]);
		Laplacian_v += yr * (v[myIndXLocal][myIndYLocal + 1] - v[myIndXLocal][myIndYLocal]);
		Laplacian_w += yr * (w[myIndXLocal][myIndYLocal + 1] - w[myIndXLocal][myIndYLocal]);			
		
		Laplacian_u = u[myIndXLocal][myIndYLocal] + parameters3var[1] * (Laplacian_u * parameters3var[14]/parameters3var[0] + reaction_u(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[11] * u[myIndXLocal][myIndYLocal]);
		Laplacian_v = v[myIndXLocal][myIndYLocal] + parameters3var[1] * (Laplacian_v * parameters3var[15]/parameters3var[0] + reaction_v(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[12] * v[myIndXLocal][myIndYLocal]);
		Laplacian_w = w[myIndXLocal][myIndYLocal] + parameters3var[1] * (Laplacian_w * parameters3var[16]/parameters3var[0] + reaction_w(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[13] * w[myIndXLocal][myIndYLocal]);

		u_new[index] = Laplacian_u > 0 ? Laplacian_u : 0;
		v_new[index] = Laplacian_v > 0 ? Laplacian_v : 0;
		w_new[index] = Laplacian_w > 0 ? Laplacian_w : 0;
	}
}

//it works in 2D only for a squared lattice
//the D texture contains the heights at each point
//the arrays u,v, and w contain the concentrations is 2D (multiplied by the height of the system)
__global__ void mainFunction2D3varBorder_D(double *u_old, double *u_new, double *v_old, double *v_new, double *w_old, double *w_new)
{
	__shared__ double u[16][16];
	__shared__ double v[16][16];
	__shared__ double w[16][16];
	int myIndX = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
	int myIndY = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
	int myIndXLocal = threadIdx.x;
	int myIndYLocal = threadIdx.y;
	int index = 0;
	bool go = false;


	if(myIndX < (dimensions[0] + 1) && myIndY < (dimensions[1] + 1))
	{
		if(myIndX >=0 && myIndX <= dimensions[0] - 1  && myIndY >= 0 && myIndY <= dimensions[1] - 1)
		{
			go = true;
		}
		if(myIndX < 0) myIndX = 0;
		if(myIndY < 0) myIndY = 0;
		if(myIndX == dimensions[0]) myIndX = dimensions[0] - 1;
		if(myIndY == dimensions[1]) myIndY  = dimensions[1] - 1;
		//global index in 1D array
		index = myIndX * dimensions[1] + myIndY;

		u[myIndXLocal][myIndYLocal] = u_old[index];
		v[myIndXLocal][myIndYLocal] = v_old[index];
		w[myIndXLocal][myIndYLocal] = w_old[index];
	}
	//solve for the indices out of domain
	__syncthreads();
	if(go && myIndXLocal > 0 && myIndXLocal < 15 && myIndYLocal > 0 && myIndYLocal < 15)
	{

		float z = tex1Dfetch(zTexture, 3 * index);
		float dzdx = tex1Dfetch(zTexture, 3 * index + 1);
		float dzdy = tex1Dfetch(zTexture, 3 * index + 2);

		
		double Laplacian_u = z * (u[myIndXLocal - 1][myIndYLocal] + u[myIndXLocal + 1][myIndYLocal] + u[myIndXLocal][myIndYLocal - 1] + u[myIndXLocal][myIndYLocal + 1] - 4 * u[myIndXLocal][myIndYLocal])/(parameters3var[0]*parameters3var[0]);
		double Laplacian_v = z * (v[myIndXLocal - 1][myIndYLocal] + v[myIndXLocal + 1][myIndYLocal] + v[myIndXLocal][myIndYLocal - 1] + v[myIndXLocal][myIndYLocal + 1] - 4 * v[myIndXLocal][myIndYLocal])/(parameters3var[0]*parameters3var[0]);
		double Laplacian_w = z * (w[myIndXLocal - 1][myIndYLocal] + w[myIndXLocal + 1][myIndYLocal] + w[myIndXLocal][myIndYLocal - 1] + w[myIndXLocal][myIndYLocal + 1] - 4 * w[myIndXLocal][myIndYLocal])/(parameters3var[0]*parameters3var[0]);	
		
		Laplacian_u += dzdx * (u[myIndXLocal + 1][myIndYLocal] - u[myIndXLocal - 1][myIndYLocal])/(2*parameters3var[0]);
		Laplacian_v += dzdx * (v[myIndXLocal + 1][myIndYLocal] - v[myIndXLocal - 1][myIndYLocal])/(2*parameters3var[0]);		
		Laplacian_w += dzdx * (w[myIndXLocal + 1][myIndYLocal] - w[myIndXLocal - 1][myIndYLocal])/(2*parameters3var[0]);		
		
		Laplacian_u += dzdy * (u[myIndXLocal][myIndYLocal + 1] - u[myIndXLocal][myIndYLocal - 1])/(2*parameters3var[0]);
		Laplacian_v += dzdy * (v[myIndXLocal][myIndYLocal + 1] - v[myIndXLocal][myIndYLocal - 1])/(2*parameters3var[0]);		
		Laplacian_w += dzdy * (w[myIndXLocal][myIndYLocal + 1] - w[myIndXLocal][myIndYLocal - 1])/(2*parameters3var[0]);		

		Laplacian_u *= parameters3var[14];
		Laplacian_v *= parameters3var[15];
		Laplacian_w *= parameters3var[16];

		
		Laplacian_u = u[myIndXLocal][myIndYLocal] + parameters3var[1]/z * (Laplacian_u + (reaction_u(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[11] * u[myIndXLocal][myIndYLocal])*z);
		Laplacian_v = v[myIndXLocal][myIndYLocal] + parameters3var[1]/z * (Laplacian_v + (reaction_v(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[12] * v[myIndXLocal][myIndYLocal])*z);
		Laplacian_w = w[myIndXLocal][myIndYLocal] + parameters3var[1]/z * (Laplacian_w + (reaction_w(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[13] * w[myIndXLocal][myIndYLocal])*z);
	
		u_new[index] = Laplacian_u;
		v_new[index] = Laplacian_v;
		w_new[index] = Laplacian_w;
	}
}

//it works in 2D only for a squared lattice
//the D texture contains the heights at each point
//the arrays u,v, and w contain the concentrations is 2D (multiplied by the height of the system)
__global__ void mainFunction2D3varBorder_D_Correction(double *u_old, double *u_new, double *v_old, double *v_new, double *w_old, double *w_new)
{
	__shared__ double u[16][16];
	__shared__ double v[16][16];
	__shared__ double w[16][16];
	int myIndX = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
	int myIndY = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
	int myIndXLocal = threadIdx.x;
	int myIndYLocal = threadIdx.y;
	int index = 0;
	bool go = false;


	if(myIndX < (dimensions[0] + 1) && myIndY < (dimensions[1] + 1))
	{
		if(myIndX >=0 && myIndX <= dimensions[0] - 1  && myIndY >= 0 && myIndY <= dimensions[1] - 1)
		{
			go = true;
		}
		if(myIndX < 0) myIndX = 0;
		if(myIndY < 0) myIndY = 0;
		if(myIndX == dimensions[0]) myIndX = dimensions[0] - 1;
		if(myIndY == dimensions[1]) myIndY  = dimensions[1] - 1;
		//global index in 1D array
		index = myIndX * dimensions[1] + myIndY;

		u[myIndXLocal][myIndYLocal] = u_old[index];
		v[myIndXLocal][myIndYLocal] = v_old[index];
		w[myIndXLocal][myIndYLocal] = w_old[index];
	}
	//solve for the indices out of domain
	__syncthreads();
	if(go && myIndXLocal > 0 && myIndXLocal < 15 && myIndYLocal > 0 && myIndYLocal < 15)
	{

		float z = tex1Dfetch(zTexture, 3 * index);
		float dzdx = tex1Dfetch(zTexture, 3 * index + 1);
		float dzdy = tex1Dfetch(zTexture, 3 * index + 2);
		float Dx = tex1Dfetch(DeffTexture, 2 * index);
		float Dy = tex1Dfetch(DeffTexture, 2 * index + 1);

		
		double Laplacian_u = Dx * (u[myIndXLocal - 1][myIndYLocal] + u[myIndXLocal + 1][myIndYLocal] - 2 * u[myIndXLocal][myIndYLocal]);
		double Laplacian_v = Dx * (v[myIndXLocal - 1][myIndYLocal] + v[myIndXLocal + 1][myIndYLocal] - 2 * v[myIndXLocal][myIndYLocal]);
		double Laplacian_w = Dx * (w[myIndXLocal - 1][myIndYLocal] + w[myIndXLocal + 1][myIndYLocal] - 2 * w[myIndXLocal][myIndYLocal]);
		
		Laplacian_u += Dy * (u[myIndXLocal][myIndYLocal - 1] + u[myIndXLocal][myIndYLocal + 1] - 2 * u[myIndXLocal][myIndYLocal]);
		Laplacian_v += Dy * (v[myIndXLocal][myIndYLocal - 1] + v[myIndXLocal][myIndYLocal + 1] - 2 * v[myIndXLocal][myIndYLocal]);
		Laplacian_w += Dy * (w[myIndXLocal][myIndYLocal - 1] + w[myIndXLocal][myIndYLocal + 1] - 2 * w[myIndXLocal][myIndYLocal]);
		
		Laplacian_u *= z/(parameters3var[0] * parameters3var[0]);
		Laplacian_v *= z/(parameters3var[0] * parameters3var[0]);
		Laplacian_w *= z/(parameters3var[0] * parameters3var[0]);
						
		Laplacian_u += dzdx * (u[myIndXLocal + 1][myIndYLocal] - u[myIndXLocal - 1][myIndYLocal])/(2*parameters3var[0]);
		Laplacian_v += dzdx * (v[myIndXLocal + 1][myIndYLocal] - v[myIndXLocal - 1][myIndYLocal])/(2*parameters3var[0]);		
		Laplacian_w += dzdx * (w[myIndXLocal + 1][myIndYLocal] - w[myIndXLocal - 1][myIndYLocal])/(2*parameters3var[0]);		
		
		Laplacian_u += dzdy * (u[myIndXLocal][myIndYLocal + 1] - u[myIndXLocal][myIndYLocal - 1])/(2*parameters3var[0]);
		Laplacian_v += dzdy * (v[myIndXLocal][myIndYLocal + 1] - v[myIndXLocal][myIndYLocal - 1])/(2*parameters3var[0]);		
		Laplacian_w += dzdy * (w[myIndXLocal][myIndYLocal + 1] - w[myIndXLocal][myIndYLocal - 1])/(2*parameters3var[0]);		

		Laplacian_u *= parameters3var[14];
		Laplacian_v *= parameters3var[15];
		Laplacian_w *= parameters3var[16];

		
		Laplacian_u = u[myIndXLocal][myIndYLocal] + parameters3var[1]/z * (Laplacian_u + (reaction_u(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[11] * u[myIndXLocal][myIndYLocal])*z);
		Laplacian_v = v[myIndXLocal][myIndYLocal] + parameters3var[1]/z * (Laplacian_v + (reaction_v(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[12] * v[myIndXLocal][myIndYLocal])*z);
		Laplacian_w = w[myIndXLocal][myIndYLocal] + parameters3var[1]/z * (Laplacian_w + (reaction_w(u[myIndXLocal][myIndYLocal], v[myIndXLocal][myIndYLocal], w[myIndXLocal][myIndYLocal]) - parameters3var[13] * w[myIndXLocal][myIndYLocal])*z);
	
		u_new[index] = Laplacian_u;
		v_new[index] = Laplacian_v;
		w_new[index] = Laplacian_w;
	}
}

void initCuda(bool random)
{
	//dimensions of the system
	int *dim = (int*)malloc(3 * sizeof(int));
	dim[0] = a;
	dim[1] = b; 
	dim[2] = c;
	hipMemcpyToSymbol(HIP_SYMBOL(dimensions), dim, 3 * sizeof(int));
	free(dim);

	pars = (double*)malloc(20 * sizeof(double));
	pars[0] = h; pars[1] = dt; pars[2] = c1; pars[3] = c2; pars[4] = c3; pars[5] = c4;
	pars[6] = c5; pars[7] = c6; pars[8] = c7;
	pars[9] = c8; pars[10] = c9; pars[11] = cu;
	pars[12] = cV; pars[13] = cw; pars[14] = Du;
	pars[15] = Dv; pars[16] = Dw; pars[17] = U;
	pars[18] = V; pars[19] = W;
	hipMemcpyToSymbol(HIP_SYMBOL(parameters3var), pars, 20 * sizeof(double));

	//set block dimensions (32,32, 1) for 2D and (16,16,4) for 3D
	if(c == 1 && constantDiffusion) //we simulate the P value at the border
	{
		DimBlock = dim3(32,32,1);
		DimGridSim = dim3(a/30 + 1 * (bool)(a % 30), b/30 + 1 * (bool)(b % 30), 1);
	}
	else
	{
		if(c > 1)
		{	
			DimBlock = dim3(16,16,4);
			DimGridSim = dim3(a/14 + 1 * (bool)(a % 14), b/14 + 1 * (bool)(b % 14), c/2 + 1 * (bool)(c % 2));				
		}
		else
		{
			DimBlock = dim3(16,16,1);
			DimGridSim = dim3(a/14 + 1 * (bool)(a % 14), b/14 + 1 * (bool)(b % 14),1);			
		}
	}

	hipMalloc((void**) &d_u_old, nodesSize*sizeof(double));
	hipMalloc((void**) &d_u_new, nodesSize*sizeof(double));
	hipMalloc((void**) &d_v_new, nodesSize*sizeof(double));
	hipMalloc((void**) &d_v_old, nodesSize*sizeof(double));
	hipMalloc((void**) &d_w_new, nodesSize*sizeof(double));
	hipMalloc((void**) &d_w_old, nodesSize*sizeof(double));
	hipMalloc((void**) &d_border, 6 * nodesSize*sizeof(float));
	if(!random)
	{
		hipMemcpy(d_u_old, u_old, nodesSize*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_v_old, v_old, nodesSize*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_w_old, w_old, nodesSize*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_u_new, u_new, nodesSize*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_v_new, v_new, nodesSize*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_w_new, w_new, nodesSize*sizeof(double), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_border, border, 6*nodesSize*sizeof(float), hipMemcpyHostToDevice);
	hipBindTexture(0, borderTexture ,d_border,6*nodesSize*sizeof(float));

	if(!constantDiffusion)
	{
		hipMalloc((void**) &d_z, 3 * nodesSize*sizeof(float));
		hipMemcpy(d_z, z, 3 * nodesSize*sizeof(float), hipMemcpyHostToDevice);
		hipBindTexture(0, zTexture ,d_z,3*nodesSize*sizeof(float));
		if(correction)
		{
			hipMalloc((void**) &d_Deff, 2 * nodesSize*sizeof(float));
			hipMemcpy(d_Deff, Deff, 2 * nodesSize*sizeof(float), hipMemcpyHostToDevice);
			hipBindTexture(0, DeffTexture ,d_Deff,2*nodesSize*sizeof(float));			
		}
	}
	cout << "Initialization done" << endl;
	hipDeviceSynchronize();


}

void cudaIteration3var_P(int number)
{
	if(number % 2 == 0)
	{
		//cout << "Starting old new" << endl;
		mainFunction3D3varBorder_P<<<DimGridSim, DimBlock>>>(d_u_old, d_u_new, d_v_old, d_v_new, d_w_old, d_w_new);
	}
	else
	{
		//cout << "Starting new old" << endl;
		mainFunction3D3varBorder_P<<<DimGridSim, DimBlock>>>(d_u_new, d_u_old, d_v_new, d_v_old, d_w_new, d_w_old);
	}
	hipDeviceSynchronize();
	//gpuErrchk( hipPeekAtLastError() );
}

void cudaIteration3var_P2D(int number)
{
	if(number % 2 == 0)
	{
		//cout << "Starting old new" << endl;
		mainFunction2D3varBorder_P<<<DimGridSim, DimBlock>>>(d_u_old, d_u_new, d_v_old, d_v_new, d_w_old, d_w_new);
	}
	else
	{
		//cout << "Starting new old" << endl;
		mainFunction2D3varBorder_P<<<DimGridSim, DimBlock>>>(d_u_new, d_u_old, d_v_new, d_v_old, d_w_new, d_w_old);
	}
	hipDeviceSynchronize();
}

void cudaIteration3var_D2D(int number)
{
	if(number % 2 == 0)
	{
		//cout << "Starting old new" << endl;
		mainFunction2D3varBorder_D<<<DimGridSim, DimBlock>>>(d_u_old, d_u_new, d_v_old, d_v_new, d_w_old, d_w_new);
	}
	else
	{
		//cout << "Starting new old" << endl;
		mainFunction2D3varBorder_D<<<DimGridSim, DimBlock>>>(d_u_new, d_u_old, d_v_new, d_v_old, d_w_new, d_w_old);
	}
	hipDeviceSynchronize();

	gpuErrchk( hipPeekAtLastError() );
}

void cudaIteration3var_D2D_Correction(int number)
{
	if(number % 2 == 0)
	{
		//cout << "Starting old new" << endl;
		mainFunction2D3varBorder_D_Correction<<<DimGridSim, DimBlock>>>(d_u_old, d_u_new, d_v_old, d_v_new, d_w_old, d_w_new);
	}
	else
	{
		//cout << "Starting new old" << endl;
		mainFunction2D3varBorder_D_Correction<<<DimGridSim, DimBlock>>>(d_u_new, d_u_old, d_v_new, d_v_old, d_w_new, d_w_old);
	}
	hipDeviceSynchronize();

	gpuErrchk( hipPeekAtLastError() );
}

void copyToHost()
{
	if(iteration % 2 == 1)
	{
		//cout << "Copy new -> new" << endl;
		hipMemcpy(u_new, d_u_new, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v_new, d_v_new, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(w_new, d_w_new, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(u_old, d_u_old, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v_old, d_v_old, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(w_old, d_w_old, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		//cout << "Copy old -> new" << endl;
		hipMemcpy(u_new, d_u_old, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v_new, d_v_old, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(w_new, d_w_old, nodesSize*sizeof(double), hipMemcpyDeviceToHost);	
		hipMemcpy(u_old, d_u_new, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v_old, d_v_new, nodesSize*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(w_old, d_w_new, nodesSize*sizeof(double), hipMemcpyDeviceToHost);		
	}
}

void changeTimeStep(double e)
{
	//increase time step for 1.5
	if(e < 0.000005)
	{
		cout << "Increasing the time step" << endl;
		pars[1] = pars[1] * 1.5;
		hipMemcpyToSymbol(HIP_SYMBOL(parameters3var), pars, 20 * sizeof(double));
	}
	else if(e > 0.00009)
	{
		cout << "Decrease the time step" << endl;
		pars[1] = pars[1] * 0.5;
		hipMemcpyToSymbol(HIP_SYMBOL(parameters3var), pars, 20 * sizeof(double));
	}
	
}

void updateMeshSpacing(double newSpacing)
{
	h = newSpacing;
	pars[0] = newSpacing;
	hipMemcpyToSymbol(HIP_SYMBOL(parameters3var), pars, 20 * sizeof(double));
	cout << "Mesh spacing updated to: " << newSpacing << endl;
}

void updateSpaceDependantDiffusion()
{
	hipMemcpy(d_z, z, 3 * nodesSize*sizeof(float), hipMemcpyHostToDevice);
	hipBindTexture(0, zTexture ,d_z,3*nodesSize*sizeof(float));

	hipMemcpy(d_Deff, Deff, 2 * nodesSize*sizeof(float), hipMemcpyHostToDevice);
	hipBindTexture(0, DeffTexture ,d_Deff,2*nodesSize*sizeof(float));			

}

void freeMemory()
{
	hipFree(d_u_old);
	hipFree(d_u_new);
	hipFree(d_v_old);
	hipFree(d_v_new);
	hipFree(d_w_old);
	hipFree(d_w_new);
	hipUnbindTexture(borderTexture);

	free(u_new);
	free(v_new);
	free(w_new);
}
