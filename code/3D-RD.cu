#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <thread>
#include <mutex>
#include <ctime>
#include <sys/stat.h>
#include <sys/types.h>

#include "loading.h"

pthread_barrier_t youCanCopyBarrier, youCanSaveBarrier;

//return true if simulation needs to be stopped
bool checkStoppingCriterion(double *e)
{
	double max = -1;
	for (int i = 0; i < a * b * c; ++i)
	{
		double e = abs(u_new[i] - u_old[i]) + abs(v_new[i] - v_old[i]) + abs(w_new[i] - w_old[i]);
		if (e > max && border[6 * i] > -2)
		{
			max = e;
		}
	}
	cout << iteration << ": " << max << endl;
	(*e) = max;
	if (max >= error)
	{
		return false;
	}
	return true;
}

//Outoput statistics at the end of the simulation
void getStats()
{
	double blackPixelsSum[3] = {0, 0, 0};
	double greenPixelsSum[3] = {0, 0, 0};
	int blackCount = 0;
	int greenCount = 0;
	for (int i = 0; i < nodesSize; ++i)
	{
		//color is black
		if (u_new[i] > v_new[i])
		{
			blackPixelsSum[0] += u_new[i];
			blackPixelsSum[1] += v_new[i];
			blackPixelsSum[2] += w_new[i];
			blackCount++;
		}
		else
		{
			greenPixelsSum[0] += u_new[i];
			greenPixelsSum[1] += v_new[i];
			greenPixelsSum[2] += w_new[i];
			greenCount++;
		}
	}
	cout << "Average concentrations black:" << endl;
	cout << blackPixelsSum[0] / blackCount << endl;
	cout << blackPixelsSum[1] / blackCount << endl;
	cout << blackPixelsSum[2] / blackCount << endl;
	cout << "Average concentrations green:" << endl;
	cout << greenPixelsSum[0] / greenCount << endl;
	cout << greenPixelsSum[1] / greenCount << endl;
	cout << greenPixelsSum[2] / greenCount << endl;
}

//save the average color in z direction as an image
void save_image_projection(char *fileName)
{
	char temp[200];
	strcpy(temp, fileName);
	strcat(temp, ".ppm");
	FILE *out = fopen(temp, "wb");
	fprintf(out, "P6 %d %d 255\n", a, b);
	for (int i = 0; i < b; ++i)
		for (int j = 0; j < a; ++j)
		{
			double sum_u = 0;
			double sum_v = 0;
			double height = 0;
			if (c > 1)
			{
				for (int k = 0; k < c; k++)
				{
					int index = k * (a * b) + j * b + i;
					if (border[6 * index] >= -1)
					{
						sum_u += u_new[index];
						sum_v += v_new[index];
						height += 1;
					}
				}
			}
			else
			{
				sum_u = u_new[j * b + i];
				sum_v = v_new[j * b + i];
				height = 1;
			}
			sum_u /= height;
			sum_v /= height;
			int G = ((sum_v - sum_u) / (sum_v + sum_u) + 0.5) * 255;
			if (G < 0)
				G = 0;
			if (G > 255)
				G = 255;
			putc(0, out);
			putc(G, out);
			putc(0, out);
		}
	fclose(out);
}

//save simulation on height k
void saveImage(char *fileName, int k)
{
	char temp[200];
	if (k >= c)
		k = c - 1;
	strcpy(temp, fileName);
	strcat(temp, ".ppm");
	FILE *out = fopen(temp, "wb");
	fprintf(out, "P6 %d %d 255\n", a, b);
	for (int i = 0; i < b; ++i)
		for (int j = 0; j < a; ++j)
		{
			int index = k * (a * b) + j * b + i;
			int R = (!(abs(border[6 * index] - 1 / h) > 0.0001) != !(abs(border[6 * index + 1] - 1 / h) > 0.0001)) && networkType != 4 ? 255 : 0;
			int G = 0;
			int B = 0;
			if (R == 0)
				R = (!(abs(border[6 * index + 2] - 1 / h) > 0.0001) != !(abs(border[6 * index + 3] - 1 / h) > 0.0001)) && networkType != 4 ? 255 : 0;
			if (border[6 * index] < -1)
			{
				R = 128;
				G = 128;
				B = 128;
			}
			else if (R == 0 && B == 0)
			{
				G = ((v_new[index] - u_new[index]) / (v_new[index] + u_new[index]) + 0.5) * 255;
				if (G < 0)
					G = 0;
				if (G > 255)
					G = 255;
			}
			putc(R, out);
			putc(G, out);
			putc(B, out);
		}
	fclose(out);
}

//save current simulation state
//if row is negative, save a s ply file
void save(char *saveFile, int row)
{
	if (row < 0)
	{
		char temp[200];
		strcpy(temp, saveFile);
		strcat(temp, ".ply");
		ofstream ply(temp);
		if (!ply)
		{
			cout << "Unable to create saving file " << saveFile << endl;
		}
		ply.precision(15);
		ply.setf(ios::fixed);
		ply.setf(ios::showpoint);
		ply << "ply \nformat ascii 1.0 \nelement vertex ";
		ply << nodesToSave;
		ply << "\nproperty double x \nproperty double y \nproperty double z";
		ply << "\nproperty uchar red  \nproperty uchar green \nproperty uchar blue\n";
		ply << "end_header\n";
		for (int i = 0; i < a; ++i)
			for (int j = 0; j < b; ++j)
				for (int k = 0; k < c; ++k)
				{
					int index = k * (a * b) + i * b + j;
					if (border[6 * index] < -1)
					{
						continue;
					}
					int value = ((v_new[index] - u_new[index]) / (v_new[index] + u_new[index]) + 0.5) * 255;
					if (value < 0)
						value = 0;
					if (value > 255)
						value = 255;
					ply << i * h;
					ply << " ";
					ply << j * h;
					ply << " ";
					ply << k * h;
					ply << " ";
					int R = 0;
					int G = 0;
					int B = 0;
					if (R == 0 && B == 0)
					{
						G = value;
					}
					ply << R;
					ply << " ";
					ply << G;
					ply << " ";
					ply << B;
					ply << endl;
				}
		ply.close();
	}
	else
	{
		saveImage(saveFile, row);
	}
}

//save current simulation state
void savePoints()
{
	bool first_iteration = true;
	while (!stop)
	{
		pthread_barrier_wait(&youCanSaveBarrier);
		cout << "Saving: " << saveIteration << endl;
		char saveFile[50];
		sprintf(saveFile, "%s/%d", saveFolder, saveIteration * saveFrequency);
		if (saveIteration == 0) //save in 3D
		{
			save(saveFile, -1);
		}
		double error;
		bool returnValue = checkStoppingCriterion(&error);
		cout << "Current: " << saveIteration * saveFrequency << endl;
		if(networkType == 0)
			save(saveFile, 0);
		else
			save(saveFile, c-1);
		if (returnValue && !first_iteration)
		{
			if (!growth)
			{
				stop = true;
				cout << "Saving thread barrier stop" << endl;
			}
			else
			{
				incrementE = true;
				first_iteration = true;
				cout << "Apply growth" << endl;
			}
			save(saveFile, -1);
		}
		else
		{
			first_iteration = false;
		}
		pthread_barrier_wait(&youCanCopyBarrier);
	}
}

void saveHexaStats()
{
	char saveFile[50];
	sprintf(saveFile, "%s/final_stats.txt", saveFolder);
	ofstream f(saveFile);
	for (int i = 0; i < centersSize; ++i)
	{
		for (int j = 0; j < 6; ++j)
		{
			f << neighbours[6 * i + j] << " ";
		}
		f << nodesPerHexa[i].size();
		for (int j = 0; j < nodesPerHexa[i].size(); ++j)
		{
			f << " " << u_new[nodesPerHexa[i][j]] << " " << v_new[nodesPerHexa[i][j]] << " " << w_new[nodesPerHexa[i][j]];
		}
		f << endl;
	}
	f.close();
}

void initHostArrays()
{
	u_new = (double *)malloc(nodesSize * sizeof(double));
	v_new = (double *)malloc(nodesSize * sizeof(double));
	w_new = (double *)malloc(nodesSize * sizeof(double));

	u_old = (double *)malloc(nodesSize * sizeof(double));
	v_old = (double *)malloc(nodesSize * sizeof(double));
	w_old = (double *)malloc(nodesSize * sizeof(double));
}

float epsilon_value(int step, float increment_factor, float initial_epsilon)
{
	return initial_epsilon + step * increment_factor;
}

int main(int argc, char **argv)
{
	if (argc < 5)
	{
		cout << "Argument 1: network type (1 - hexagonal prisms, 0 - gaussian bumps)" << endl;
		cout << "Argument 2: initial conditions if network type is 1 (1 - uniformly colored scales, 0 - randomized uniform steady state)" << endl;
		cout << "Argument 2: gaussian bumps sigma (integer) if network type is 0" << endl;
		cout << "Argument 3: total height of the simulation domain" << endl;
		cout << "Argument 4: domain thickness between prisms/gaussian bumps" << endl;
		cout << "Argument 5: mesh spacing epsilon" << endl;
		exit(-1);
	}
	h = atof(argv[5]);
	periodic = true;
	bool initialCond = atoi(argv[2]);
	border_thickness_in_elements = 1 / h;
	cout << "Element spacing: " << h << endl;
	cout << "Border thickness in elements: " << border_thickness_in_elements << endl;
	bt = 0.5 * border_thickness_in_elements + 1; //border thickess (for border thickness of x elements put 0.5*x + 1)
	dt = 0.012 * (h * h);
	error = 0.0000001;
	saveFrequency = 10000;
	constantDiffusion = true;
	//load network
	int res = 200;
	cout << "Resolution and spacing" << endl;
	cout << res << " " << h << endl;
	networkType = atoi(argv[1]);
	int maxZ = atoi(argv[3]);
	int sigma = -1;
	if(networkType == 0)
		sigma = atoi(argv[2]);
	int type = 1;
	if (networkType == 1) //hexagonal prisms
	{
		int freeZ = atoi(argv[4]);
		P = 0;
		c = maxZ;
		type = loadRegularNetwork("../networks/hexa10by10.txt", 1, 1, res);
		if (bt < 1.5)
		{
			getPixels(type, maxZ, freeZ); //one element at the border
			nodesToSave = nodesSize;
		}
		else
		{
			nodesToSave = getPixelsP0btMoreThanOne(type, maxZ, freeZ);
		}
		sprintf(saveFolder, "../output/prisms_%d_%d_%d_%s", initialCond, freeZ, maxZ, argv[5]);
		int result = mkdir(saveFolder, S_IRUSR | S_IWUSR | S_IXUSR);
		nodesSize = a * b * c;
		cout << "Nodes size" << nodesSize << endl;
	}
	else if (networkType == 0)  //gaussian bumps
	{
		if(sigma < 0)
			cout << "Wrong sigma value" << endl;
		int freeZ = atoi(argv[4]);
		char gaussNetworkFile[200];
		sprintf(gaussNetworkFile, "../networks/gauss10by10_%d_%d_%d.txt", freeZ, maxZ, sigma);
		P = 0;
		periodic = false;
		nodesToSave = getPixelsGaussianBumps(gaussNetworkFile);
		nodesSize = a * b * c;
		sprintf(saveFolder, "../output/gauss_%d_%d_%d_%s", freeZ, maxZ, sigma, argv[5]);
		int result = mkdir(saveFolder, S_IRUSR | S_IWUSR | S_IXUSR);
	}
	else
	{
		cout << "Wrong network type" << endl;
		exit(-1);
	}
	//just in case
	correctBordersToTheNonsimulatedNodes();
	if (type == 1 && periodic)
		hexaNetworkCorrection();

	cout << "Dimensions:" << a << " " << b << " " << c << " " << endl;
	cout << "Number of nodes: " << nodesSize << endl;
	cout << "Number of nodes to save: " << nodesToSave << endl;

	//simulation parameters
	c1 = -0.04;
	c2 = -0.056;
	c3 = 0.382;
	c4 = -0.05;
	c5 = 0;
	c6 = 0.25;
	c7 = 0.016;
	c8 = -0.03;
	c9 = 0.24;
	cu = 0.02;
	cV = 0.025;
	cw = 0.06;

	Du = 1.125;
	Dv = 1.125;
	Dw = 12 * Du;

	cout << "Diffusion coeff " << Du << endl;

	U = 0.5;
	V = 0.5;
	W = 0.5;

	if(initialCond && networkType == 1)
	{
		initialGreen[0] = 1.2;
		initialGreen[1] = 6.6;
		initialGreen[2] = 2.3;
		initialBlack[0] = 5.3;
		initialBlack[1] = 0.92;
		initialBlack[2] = 4;
	}
	else
	{
		initialUniform[0] = 3.47561;
		initialUniform[1] = 3.0488;
		initialUniform[2] = 3.40243;
		initialUniform[0] = 3.47561;
		initialUniform[1] = 3.0488;
		initialUniform[2] = 3.40243;
	}

	double randomIntervalSize = 0.1;
	initHostArrays();
		//can be replaced by loadInitialConcentrations to continue a simulation
		//bool ret = loadInitialConcentrations("/home/lane/PROJECTS/RD/code/concentrations_7.txt");
	if(networkType == 1)
		bool ret = loadInitialColors("../networks/hexa10by10_c.txt", randomIntervalSize, type, -1);
	else
		createRandomInitialConditions(randomIntervalSize);

	cout << "Initializing cuda" << endl;
	initCuda(false);
	iteration = 0;
	stop = false;
	saving = true;

	pthread_barrier_init(&youCanCopyBarrier, NULL, 2);
	pthread_barrier_init(&youCanSaveBarrier, NULL, 2);
	thread savingThread(savePoints);
	copyToHost();
	saveIteration = 0;
	pthread_barrier_wait(&youCanSaveBarrier);

	cout << "Starting the simulation" << endl;

	if (c == 1)
	{
		cout << "Running 2D version" << endl;
		while (1)
		{
			cout << "Thread 0: " << iteration << endl;
			for (int i = 0; i < saveFrequency; ++i)
			{
				cudaIteration3var_P2D(iteration);
				iteration++;
			}
			pthread_barrier_wait(&youCanCopyBarrier);
			if (stop)
			{
				cout << "Done" << endl;
				break;
			}
			copyToHost();
			saveIteration++;
			pthread_barrier_wait(&youCanSaveBarrier);
		}
	}
	else
	{
		cout << "Running 3D version" << endl;
		while (1)
		{
			cout << "Thread 0: " << iteration << endl;
			for (int i = 0; i < saveFrequency; ++i)
			{
				cudaIteration3var_P(iteration);
				iteration++;
			}
			pthread_barrier_wait(&youCanCopyBarrier);
			if (stop)
			{
				cout << "Done" << endl;
				break;
			}
			copyToHost();
			saveIteration++;
			pthread_barrier_wait(&youCanSaveBarrier);
		}
	}
	savingThread.join();
	return 0;
}